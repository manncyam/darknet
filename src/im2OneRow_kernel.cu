#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "im2OneRow.h"

// convert rgb to 1D vector with all values of r, g and b in order
__global__ void im2OneRow_kernel(unsigned char* dev_src, float* dev_dst, int channels, int rows, int cols, int step)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	int k = blockIdx.y;

	int index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x;
	int rgb_index = i * step + j * channels + k;

	dev_dst[index] = dev_src[rgb_index] / 255.0f;
}

// used to launch kernel code
void im2OneRow(unsigned char* dev_src, float* dev_dst, int channels, int rows, int cols, int step)
{
	dim3 dg(rows, channels, 1);
	dim3 db(cols, 1, 1);
	// Launch a kernel on the GPU with one thread for each element.
	im2OneRow_kernel << < dg, db >> >(dev_src, dev_dst, channels, rows, cols, step);
}